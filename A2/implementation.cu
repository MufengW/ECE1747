#include "hip/hip_runtime.h"
#include "implementation.h"
#include "util_gpu_err_check.cuh"
#include <iostream>
#include "stdio.h"
#include <hip/hip_runtime.h>

#define MAX_THREADS_PER_BLOCK 256
#define BLOCKSZ (MAX_THREADS_PER_BLOCK * 2)
#define NUM_BANKS 16
#define LOG_NUM_BANKS 4
#define CONFLICT_FREE_OFFSET(n) ((n) >> NUM_BANKS + ((n) >> (2 * LOG_NUM_BANKS)))
#define MAX_SHARE_SIZE (BLOCKSZ + CONFLICT_FREE_OFFSET(BLOCKSZ - 1))

void printSubmissionInfo()
{
    // This will be published in the leaderboard on piazza
    // Please modify this field with something interesting
    char nick_name[] = "default-team";

    // Please fill in your information (for marking purposes only)
    char student_first_name[] = "John";
    char student_last_name[] = "Doe";
    char student_student_number[] = "00000000";

    // Printing out team information
    printf("*******************************************************************************************************\n");
    printf("Submission Information:\n");
    printf("\tnick_name: %s\n", nick_name);
    printf("\tstudent_first_name: %s\n", student_first_name);
    printf("\tstudent_last_name: %s\n", student_last_name);
    printf("\tstudent_student_number: %s\n", student_student_number);
}

__device__ int32_t device_newIdxWithBCAO(int32_t idx) {
    return idx + CONFLICT_FREE_OFFSET(idx);
}

void debug_print_list(int32_t* list, size_t size) {
    int32_t* h_list = new int32_t[size];
    hipMemcpy(h_list, list, size * sizeof(int32_t), hipMemcpyDeviceToHost);
    for (size_t i = 0; i < size; ++i) {
        printf("%u:%d ", i, h_list[i]);
    }
    std::cout << std::endl << std::endl;
    delete[] h_list;
}

__device__ void device_upSweep(int32_t *sharedData, int32_t threadId, int32_t totalLeaves, bool BCAO) {
    int32_t offset = 1;
    for (int32_t depth = totalLeaves >> 1; depth > 0; depth >>= 1) {
        if (threadId < depth) {
            int32_t leftIndex = offset * (2 * threadId + 1) - 1;
            int32_t rightIndex = offset * (2 * threadId + 2) - 1;

            if (BCAO) {
                sharedData[device_newIdxWithBCAO(rightIndex)] += sharedData[device_newIdxWithBCAO(leftIndex)];
            } else {
                sharedData[rightIndex] += sharedData[leftIndex];
            }
        }
        offset *= 2;
        __syncthreads();
    }
}

__device__ void device_downSweep(int32_t *sharedData, int32_t threadId, int32_t totalLeaves,
    int32_t *blockSums, int32_t blockId, bool BCAO) {

    for (unsigned int stride = totalLeaves >> 1; stride > 0; stride >>= 1) {
        int index = (threadId + 1) * stride * 2 - 1;
        if (index + stride < totalLeaves) {
            if (BCAO) {
                sharedData[device_newIdxWithBCAO(index + stride)] += sharedData[device_newIdxWithBCAO(index)];
            } else {
                sharedData[index + stride] += sharedData[index];
            }
        }
        __syncthreads();
    }
    if (threadId == 0) {
        int lastIdx = BCAO ? device_newIdxWithBCAO(totalLeaves - 1) : totalLeaves - 1;
        blockSums[blockId] = sharedData[lastIdx];
    }
}

__global__ void kernel_addPrefixSums(int32_t *prefixSum, int32_t *blockValues, size_t numElements)
{
    int threadId = threadIdx.x;
    int blockId = blockIdx.x;
    int blockOffset = (blockId+1) * BLOCKSZ;
    int leftIndex = threadId * 2 + blockOffset;
    int rightIndex = threadId * 2 + 1 + blockOffset;

    int32_t valueToAdd = blockValues[blockId];
    // Add the value to the elements of the prefix sum array
    if (leftIndex < numElements) {
        prefixSum[leftIndex] += valueToAdd;
    }
    if (rightIndex < numElements) {
        prefixSum[rightIndex] += valueToAdd;
    }
}

__global__ void kernel_parallelLargeScan(int32_t *data, int32_t *prefixSum, size_t N, int32_t *blockSums, bool BCAO) {

    __shared__ int32_t sharedPrefixSum[BLOCKSZ];
    int threadId = threadIdx.x;
    int blockId = blockIdx.x;
    int blockOffset = blockId * BLOCKSZ;
    int leafNum = BLOCKSZ;

    // Load data into shared memory
    if (BCAO) {
        int leftIndex = threadId;
        int rightIndex = threadId + (leafNum / 2);
        sharedPrefixSum[device_newIdxWithBCAO(leftIndex)] =
            (leftIndex + blockOffset < N) ? data[leftIndex + blockOffset] : 0;
        sharedPrefixSum[device_newIdxWithBCAO(rightIndex)] =
            (rightIndex + blockOffset < N) ? data[rightIndex + blockOffset] : 0;
    } else {
        sharedPrefixSum[threadId * 2] =
            (threadId * 2 + blockOffset < N) ? data[threadId * 2 + blockOffset] : 0;
        sharedPrefixSum[threadId * 2 + 1] =
            (threadId * 2 + 1 + blockOffset < N) ? data[threadId * 2 + 1 + blockOffset] : 0;
    }
    __syncthreads();

    device_upSweep(sharedPrefixSum, threadId, leafNum, BCAO);
    device_downSweep(sharedPrefixSum, threadId, leafNum, blockSums, blockId, BCAO);

    // Writing the results back to global memory
    if (BCAO) {
        if (threadId * 2 + blockOffset < N) {
            prefixSum[threadId * 2 + blockOffset] = sharedPrefixSum[device_newIdxWithBCAO(threadId * 2)];
        }
        if (threadId * 2 + 1 + blockOffset < N) {
            prefixSum[threadId * 2 + 1 + blockOffset] = sharedPrefixSum[device_newIdxWithBCAO(threadId * 2 + 1)];
        }
    } else {
        if (threadId * 2 + blockOffset < N) {
            prefixSum[threadId * 2 + blockOffset] = sharedPrefixSum[threadId * 2];
        }
        if (threadId * 2 + 1 + blockOffset < N) {
            prefixSum[threadId * 2 + 1 + blockOffset] = sharedPrefixSum[threadId * 2 + 1];
        }
    }
    __syncthreads();
}

void parallelPrefixSumLargeData(int32_t *d_data, int32_t *d_prefixSum, size_t size) {
    size_t blockNum = (size + BLOCKSZ - 1) / BLOCKSZ;

    int32_t *d_blockSums, *d_blockSumsPrefix;
    hipMalloc(&d_blockSums, blockNum * sizeof(int32_t));
    hipMalloc(&d_blockSumsPrefix, blockNum * sizeof(int32_t));

    // Bank Conflict Avoidance Optimization
    bool BCAO = false;
    kernel_parallelLargeScan<<<blockNum, MAX_THREADS_PER_BLOCK>>>(d_data, d_prefixSum, size, d_blockSums, BCAO);

    if (blockNum > 1) {
        parallelPrefixSumLargeData(d_blockSums, d_blockSumsPrefix, blockNum);
        kernel_addPrefixSums<<<blockNum-1, MAX_THREADS_PER_BLOCK>>>(d_prefixSum, d_blockSumsPrefix, size);
    }

    hipFree(d_blockSums);
    hipFree(d_blockSumsPrefix);
}

void parallelPrefixSumLargeData2(int32_t *d_data, int32_t *d_prefixSum, size_t size) {
    size_t blockNum = (size + BLOCKSZ - 1) / BLOCKSZ;

    int32_t *d_blockSums;
    hipMalloc(&d_blockSums, blockNum * sizeof(int32_t));

    bool BCAO = false;

    // Step 1: Perform the scan on each block
    kernel_parallelLargeScan<<<blockNum, MAX_THREADS_PER_BLOCK>>>(d_data, d_prefixSum, size, d_blockSums, BCAO);
    // debug_print_list(d_prefixSum, size);

    // Handling the case when there are multiple blocks
    if (blockNum > 1) {
        // Step 2: Iteratively compute the scan of the block sums
        size_t currentSize = blockNum;
        int32_t *d_tempBlockSums, *d_swap;
        hipMalloc(&d_tempBlockSums, blockNum * sizeof(int32_t));

        // Copy the initial block sums to the temporary array
        hipMemcpy(d_tempBlockSums, d_blockSums, blockNum * sizeof(int32_t), hipMemcpyDeviceToDevice);

        while (currentSize > 1) {
            size_t currentBlockNum = (currentSize + BLOCKSZ - 1) / BLOCKSZ;

            kernel_parallelLargeScan<<<currentBlockNum, MAX_THREADS_PER_BLOCK>>>(d_tempBlockSums, d_blockSums, currentSize, d_blockSums, BCAO);

            // Swap the pointers for the next iteration
            d_swap = d_tempBlockSums;
            d_tempBlockSums = d_blockSums;
            d_blockSums = d_swap;

            currentSize = currentBlockNum;
        }

        // Step 3: Add the scanned block sums to the prefix sum of each block
        // Be sure to use the correct array that holds the final scanned block sums
        kernel_addPrefixSums<<<blockNum - 1, MAX_THREADS_PER_BLOCK>>>(d_prefixSum, d_tempBlockSums, size);

        hipFree(d_tempBlockSums);
    }

    hipFree(d_blockSums);
}

void implementation(const int32_t *d_input, int32_t *d_output, size_t size) {
    // Perform the recursive scan
    parallelPrefixSumLargeData(const_cast<int32_t*>(d_input), d_output, size);
}
