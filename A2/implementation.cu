#include "hip/hip_runtime.h"
#include "implementation.h"
#include "util_gpu_err_check.cuh"
#include <iostream>
#include "stdio.h"
#include <hip/hip_runtime.h>

#define THREADS_PER_BLOCK 128
#define BLOCK_SIZE (THREADS_PER_BLOCK * 2)
#define NUM_BANKS 16
#define LOG_NUM_BANKS 4
#define CONFLICT_FREE_OFFSET(n) ((n) >> NUM_BANKS + ((n) >> (2 * LOG_NUM_BANKS)))
#define MAX_SHARE_SIZE (BLOCK_SIZE + CONFLICT_FREE_OFFSET(BLOCK_SIZE - 1))

void printSubmissionInfo()
{
    // This will be published in the leaderboard on piazza
    // Please modify this field with something interesting
    char nick_name[] = "anonymous";

    // Please fill in your information (for marking purposes only)
    char student_first_name[] = "Mufeng";
    char student_last_name[] = "Wang";
    char student_student_number[] = "1003804467";

    // Printing out team information
    printf("*******************************************************************************************************\n");
    printf("Submission Information:\n");
    printf("\tnick_name: %s\n", nick_name);
    printf("\tstudent_first_name: %s\n", student_first_name);
    printf("\tstudent_last_name: %s\n", student_last_name);
    printf("\tstudent_student_number: %s\n", student_student_number);
}

__device__ int32_t device_newIdxWithBCAO(int32_t idx) {
    return idx + CONFLICT_FREE_OFFSET(idx);
}

__device__ void device_upSweep(int32_t *sharedData, int32_t threadId, int32_t totalLeaves, bool BCAO) {
    int32_t offset = 1;
    for (int32_t depth = totalLeaves >> 1; depth > 0; depth >>= 1) {
        if (threadId < depth) {
            int32_t leftIndex = offset * (2 * threadId + 1) - 1;
            int32_t rightIndex = offset * (2 * threadId + 2) - 1;

            if (BCAO) {
                sharedData[device_newIdxWithBCAO(rightIndex)] += sharedData[device_newIdxWithBCAO(leftIndex)];
            } else {
                sharedData[rightIndex] += sharedData[leftIndex];
            }
        }
        offset *= 2;
        __syncthreads();
    }
}

__device__ void device_downSweep(int32_t *sharedData, int32_t threadId, int32_t totalLeaves,
    int32_t *blockSums, int32_t blockId, bool BCAO) {

    for (unsigned int stride = totalLeaves >> 1; stride > 0; stride >>= 1) {
        int index = (threadId + 1) * stride * 2 - 1;
        if (index + stride < totalLeaves) {
            if (BCAO) {
                sharedData[device_newIdxWithBCAO(index + stride)] += sharedData[device_newIdxWithBCAO(index)];
            } else {
                sharedData[index + stride] += sharedData[index];
            }
        }
        __syncthreads();
    }
    if (threadId == 0) {
        int lastIdx = BCAO ? device_newIdxWithBCAO(totalLeaves - 1) : totalLeaves - 1;
        blockSums[blockId] = sharedData[lastIdx];
    }
}

__global__ void kernel_addPrefixSums(int32_t *prefixSum, int32_t *blockValues, size_t numElements)
{
    int threadId = threadIdx.x;
    int blockId = blockIdx.x;
    int blockOffset = (blockId+1) * BLOCK_SIZE;
    int leftIndex = threadId * 2 + blockOffset;
    int rightIndex = threadId * 2 + 1 + blockOffset;

    int32_t valueToAdd = blockValues[blockId];

    if (leftIndex < numElements) {
        prefixSum[leftIndex] += valueToAdd;
    }
    if (rightIndex < numElements) {
        prefixSum[rightIndex] += valueToAdd;
    }
}

__global__ void kernel_parallelLargeScan(int32_t *data, int32_t *prefixSum,
    size_t size, int32_t *blockSums, bool BCAO) {
    extern __shared__ int32_t sharedPrefixSum[];
    int threadId = threadIdx.x;
    int blockId = blockIdx.x;
    int blockOffset = blockId * BLOCK_SIZE;
    int leafNum = BLOCK_SIZE;

    // Load data into shared memory
    if (BCAO) {
        int leftIndex = threadId;
        int rightIndex = threadId + (leafNum / 2);
        sharedPrefixSum[device_newIdxWithBCAO(leftIndex)] =
            (leftIndex + blockOffset < size) ? data[leftIndex + blockOffset] : 0;
        sharedPrefixSum[device_newIdxWithBCAO(rightIndex)] =
            (rightIndex + blockOffset < size) ? data[rightIndex + blockOffset] : 0;
    } else {
        sharedPrefixSum[threadId * 2] =
            (threadId * 2 + blockOffset < size) ? data[threadId * 2 + blockOffset] : 0;
        sharedPrefixSum[threadId * 2 + 1] =
            (threadId * 2 + 1 + blockOffset < size) ? data[threadId * 2 + 1 + blockOffset] : 0;
    }
    __syncthreads();

    device_upSweep(sharedPrefixSum, threadId, leafNum, BCAO);
    device_downSweep(sharedPrefixSum, threadId, leafNum, blockSums, blockId, BCAO);

    // Writing the results back to global memory
    if (BCAO) {
        if (threadId * 2 + blockOffset < size) {
            prefixSum[threadId * 2 + blockOffset] = sharedPrefixSum[device_newIdxWithBCAO(threadId * 2)];
        }
        if (threadId * 2 + 1 + blockOffset < size) {
            prefixSum[threadId * 2 + 1 + blockOffset] = sharedPrefixSum[device_newIdxWithBCAO(threadId * 2 + 1)];
        }
    } else {
        if (threadId * 2 + blockOffset < size) {
            prefixSum[threadId * 2 + blockOffset] = sharedPrefixSum[threadId * 2];
        }
        if (threadId * 2 + 1 + blockOffset < size) {
            prefixSum[threadId * 2 + 1 + blockOffset] = sharedPrefixSum[threadId * 2 + 1];
        }
    }
    __syncthreads();
}

void parallelPrefixSumLargeData(int32_t *d_data, int32_t *d_prefixSum, size_t size) {
    size_t blockNum = (size + BLOCK_SIZE - 1) / BLOCK_SIZE;

    int32_t *d_blockSums, *d_blockSumsPrefix;
    hipMalloc(&d_blockSums, blockNum * sizeof(int32_t));

    // Bank Conflict Avoidance Optimization
    bool BCAO = true;
    size_t shared_size = BCAO ? MAX_SHARE_SIZE : BLOCK_SIZE;
    kernel_parallelLargeScan<<<blockNum, THREADS_PER_BLOCK, shared_size * sizeof(int32_t)>>>
        (d_data, d_prefixSum, size, d_blockSums, BCAO);

    if (blockNum > 1) {
        hipMalloc(&d_blockSumsPrefix, blockNum * sizeof(int32_t));
        parallelPrefixSumLargeData(d_blockSums, d_blockSumsPrefix, blockNum);
        kernel_addPrefixSums<<<blockNum - 1, THREADS_PER_BLOCK>>>(d_prefixSum, d_blockSumsPrefix, size);
        hipFree(d_blockSumsPrefix);
    }
    hipFree(d_blockSums);
}

void implementation(const int32_t *d_input, int32_t *d_output, size_t size) {
    // Perform the recursive scan
    parallelPrefixSumLargeData(const_cast<int32_t*>(d_input), d_output, size);
}
